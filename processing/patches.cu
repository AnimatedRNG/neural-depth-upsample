#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_vector_types.h"

__device__ float rand(float x, float y){
    //return fract(sin(dot(co.xy ,vec2(12.9898,78.233))) * 43758.5453);
    float a = sin(x * 12.9898 + y * 78.233) * 43758.5453;
    return a - floor(a);
}

extern "C"
__global__ void image_hash(
    float* image,
    float* high_res_image,
    unsigned int width,
    unsigned int height,
    float* patches,
    float* results,
    unsigned int patches_length,
    unsigned int* atomic_counter) {
    const uint center_i = threadIdx.x + blockIdx.x * blockDim.x;
    const uint center_j = threadIdx.y + blockIdx.y * blockDim.y;
    const uint center_k = threadIdx.z + blockIdx.z * blockDim.z;
    
    const uint global_id = center_i * height + center_j;
    
    float patch[7][7][4];
    float high_res[2][2][3];
    unsigned int num_nonzero = 0;

    // Very inefficient
    // TODO: Move brightness scaling and color channel swap to a python
    // class that acts like keras ImageDataGenerator
    int ordering[3];
    int start = global_id % 3;
    int a = 0;
    for (int i = start; i < start + 3; i++) {
        ordering[a++] = i % 3;
    }

    float brightness = 1.0 + rand((float) center_i, (float) center_j) * 0.2;
    
    for (int offset_i = 0; offset_i < 2; offset_i++) {
        for (int offset_j = 0; offset_j < 2; offset_j++) {
            int i = (center_i * 2) + offset_i;
            int j = (center_j * 2) + offset_j;
            int ind = 3 * (i * (height * 2) + j);
            
            high_res[offset_i][offset_j][0] =
                high_res_image[ind + ordering[0]] * brightness;
            high_res[offset_i][offset_j][1] =
                high_res_image[ind + ordering[1]] * brightness;
            high_res[offset_i][offset_j][2] =
                high_res_image[ind + ordering[2]] * brightness;
        }
    }
    
    for (int offset_i = -3; offset_i <= 3; offset_i++) {
        for (int offset_j = -3; offset_j <= 3; offset_j++) {
            int i = offset_i + center_i;
            int j = offset_j + center_j;
            
            int norm_offset_i = offset_i + 3;
            int norm_offset_j = offset_j + 3;
            
            float r = 0.0;
            float g = 0.0;
            float b = 0.0;
            float d = 0.0;
            int image_index = 0;
            if (i >= 0 && j >= 0 && i < width && i < height) {
                image_index = 4 * (i * height + j);
                r = image[image_index + ordering[0]] * brightness;
                g = image[image_index + ordering[1]] * brightness;
                b = image[image_index + ordering[2]] * brightness;
                d = image[image_index + 3];
                num_nonzero++;
            }
            
            patch[norm_offset_i][norm_offset_j][0] = r;
            patch[norm_offset_i][norm_offset_j][1] = g;
            patch[norm_offset_i][norm_offset_j][2] = b;
            patch[norm_offset_i][norm_offset_j][3] = d;
        }
    }

    unsigned int my_patch_id = atomicAdd(atomic_counter, 1);
    for (int offset_i = -3; offset_i <= 3; offset_i++) {
        for (int offset_j = -3; offset_j <= 3; offset_j++) {
            int norm_offset_i = offset_i + 3;
            int norm_offset_j = offset_j + 3;
            unsigned int patches_index = my_patch_id * (7 * 7 * 4) +
                                         norm_offset_i * (7 * 4) + norm_offset_j * 4;

            patches[patches_index] = patch[norm_offset_i][norm_offset_j][0];
            patches[patches_index + 1] = patch[norm_offset_i][norm_offset_j][1];
            patches[patches_index + 2] = patch[norm_offset_i][norm_offset_j][2];
            patches[patches_index + 3] = patch[norm_offset_i][norm_offset_j][3];
        }
    }

    for (int offset_i = 0; offset_i < 2; offset_i++) {
        for (int offset_j = 0; offset_j < 2; offset_j++) {
            unsigned int results_index =
                my_patch_id * (2 * 2 * 3) + offset_i * (2 * 3) + offset_j * 3;
            results[results_index] = high_res[offset_i][offset_j][0];
            results[results_index + 1] = high_res[offset_i][offset_j][1];
            results[results_index + 2] = high_res[offset_i][offset_j][2];
        }
    }
}
