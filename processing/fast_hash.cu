#include "hip/hip_runtime.h"
#include "stdio.h"
#include "hip/hip_vector_types.h"

extern "C"
__global__ void image_hash(
    float* image,
    float* high_res_image,
    unsigned int width,
    unsigned int height,
    unsigned long long* hash_table,
    unsigned int hash_table_length,
    unsigned int hash_table_width,
    float* patches,
    float* results,
    unsigned int patches_length, unsigned int* atomic_counter) {
    const uint center_i = threadIdx.x + blockIdx.x * blockDim.x;
    const uint center_j = threadIdx.y + blockIdx.y * blockDim.y;
    const uint center_k = threadIdx.z + blockIdx.z * blockDim.z;
    
    const uint global_id = center_i * height + center_j;
    
    float patch[7][7][4];
    float high_res[2][2][3];
    float luma[7][7];
    float average = 0.0;
    unsigned int num_nonzero = 0;

    for (int offset_i = 0; offset_i < 2; offset_i++) {
        for (int offset_j = 0; offset_j < 2; offset_j++) {
            int i = (center_i * 2) + offset_i;
            int j = (center_j * 2) + offset_j;
            int ind = 3 * (i * (height * 2) + j);
            
            high_res[offset_i][offset_j][0] =
                high_res_image[ind];
            high_res[offset_i][offset_j][1] =
                high_res_image[ind + 1];
            high_res[offset_i][offset_j][2] =
                high_res_image[ind + 2];
        }
    }
    
    for (int offset_i = -3; offset_i <= 3; offset_i++) {
        for (int offset_j = -3; offset_j <= 3; offset_j++) {
            int i = offset_i + center_i;
            int j = offset_j + center_j;
            
            int norm_offset_i = offset_i + 3;
            int norm_offset_j = offset_j + 3;
            
            float r = 0.0;
            float g = 0.0;
            float b = 0.0;
            float d = 0.0;
            int image_index = 0;
            if (i >= 0 && j >= 0 && i < width && i < height) {
                image_index = 4 * (i * height + j);
                r = image[image_index];
                g = image[image_index + 1];
                b = image[image_index + 2];
                d = image[image_index + 3];
                num_nonzero++;
            }
            
            patch[norm_offset_i][norm_offset_j][0] = r;
            patch[norm_offset_i][norm_offset_j][1] = g;
            patch[norm_offset_i][norm_offset_j][2] = b;
            patch[norm_offset_i][norm_offset_j][3] = d;
            
            float this_luma = sqrt(0.2061 * r * r +
                                   0.5283 * g * g +
                                   0.1026 * b * b +
                                   0.1 * d * d);
                                   
            luma[norm_offset_i][norm_offset_j] = this_luma;
            average += this_luma;
        }
    }
    
    average /= num_nonzero;
    
    unsigned long long hash_value = 0;
    int current_index = 0;
    for (int offset_i = -3; offset_i <= 3; offset_i++) {
        for (int offset_j = -3; offset_j <= 3; offset_j++) {
            int norm_offset_i = offset_i + 3;
            int norm_offset_j = offset_j + 3;
            
            float this_luma = luma[norm_offset_i][norm_offset_j];
            if (this_luma > average) {
                hash_value |= 1ULL << current_index;
            }
            current_index++;
        }
    }
    
    unsigned int hash_base_index = (hash_value % hash_table_length) *
                                   hash_table_width * 2;
    for (int a = 0; a < hash_table_width * 2; a += 2) {
        unsigned int hash_index = hash_base_index + a;
        
        // Only replaces if there wasn't anything already there...
        // potentially inefficient
        if (atomicCAS(&(hash_table[hash_index]), 0, hash_value) == 0) {
            unsigned int my_patch_id = atomicAdd(atomic_counter, 1);
            hash_table[hash_index + 1] = my_patch_id;
            
            for (int offset_i = -3; offset_i <= 3; offset_i++) {
                for (int offset_j = -3; offset_j <= 3; offset_j++) {
                    int norm_offset_i = offset_i + 3;
                    int norm_offset_j = offset_j + 3;
                    unsigned int patches_index = my_patch_id * (7 * 7 * 4) +
                                                 norm_offset_i * (7 * 4) + norm_offset_j * 4;
                    
                    if (patches_index < 20000000) {
                        patches[patches_index] = patch[norm_offset_i][norm_offset_j][0];
                        patches[patches_index + 1] = patch[norm_offset_i][norm_offset_j][1];
                        patches[patches_index + 2] = patch[norm_offset_i][norm_offset_j][2];
                        patches[patches_index + 3] = patch[norm_offset_i][norm_offset_j][3];
                    }
                }
            }

            for (int offset_i = 0; offset_i < 2; offset_i++) {
                for (int offset_j = 0; offset_j < 2; offset_j++) {
                    unsigned int results_index =
                        my_patch_id * (2 * 2 * 3) + offset_i * (2 * 3) + offset_j * 3;
                    if (results_index < 20000000) {
                        results[results_index] = high_res[offset_i][offset_j][0];
                        results[results_index + 1] = high_res[offset_i][offset_j][1];
                        results[results_index + 2] = high_res[offset_i][offset_j][2];
                    }
                }
            }
            
            break;
        }
    }
    
    image[4 * global_id] = luma[3][3];
    image[4 * global_id + 1] = luma[3][3];
    image[4 * global_id + 2] = luma[3][3];
    image[4 * global_id + 3] = luma[3][3];
}
